/*

    CS-550 (Fall-2021)

    Assignment-7 : Question 1 Problem 1

    Author:

        Nazmul (mh2752@nau.edu)

*/



#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "omp.h"
#include <iostream>
using namespace std;



// **********************************************************  Macro & Misc. Declarations ************************************************************


        // Number of datapoints in the
        // test dataset:
        #define N 100000

        // Seed for the random data
        // generator:
        #define SEED 72

        // For GPU implementation
        #define BLOCKSIZE 1024


        // Typedef for point data 
        // structure:
        typedef struct pointData
        {
            double x;
            double y;
        } pointData;


        // ------------------------------------------------- Prototype of data generation function ------------------------------------------
        void generateDataset(struct pointData * data);


        // ------------------------------------------------- Prototype of the function for warming up the GPU -------------------------------
        void warmUpGPU();

        // ------------------------------------------------- Prototype of the Kernel Function -----------------------------------------------
        __global__ void pointEpsilonBruteForce(pointData* all_data_points,int *device_points_within_epsilon,double *device_epsilon);



// *******************************************************************************************************************************************


int main(int argc, char *argv[])
{
	
	
	//Read epsilon distance from command line
	if (argc!=2)
	{
        printf("\n Incorrect number of input parameters. Please input an epsilon distance.\n");
        return 0;
	}
	
	
	char inputEpsilon[20];
	strcpy(inputEpsilon,argv[1]);
	double epsilon=atof(inputEpsilon);
	
	

	//generate dataset:
	pointData * data;
	data=(pointData*)malloc(sizeof(pointData)*N);
	printf("\nSize of dataset (MiB): %f",(2.0*sizeof(double)*N*1.0)/(1024.0*1024.0));
	generateDataset(data);


	omp_set_num_threads(1);

	
    // Overall starting timestamp
    double tstart=omp_get_wtime();
	

	//Write your code here:
	//The data you need to use is stored in the variable "data", 
	//which is of type pointData:


        // Variale to store the total number of points within
        // the epsilon on the CPU:
        int hpwe;
        int *host_points_within_epsilon = &hpwe;
        *host_points_within_epsilon = 0;

        // Variable storing the count of the total number
        // of points within epsilon on the GPU:
        int *device_points_within_epsilon;

        // Array on the GPU for holding
        // the pointData objects:
        pointData *device_data;

        // Varaible for epsilon value
        // on the GPU:
        double *device_epsilon;


        // ----------------- Allocate memories on the GPU ------------------------------
        
            
            // Warming up the GPU first:
            warmUpGPU();           
            
            
            hipError_t errorCode = hipSuccess;
            
            
            
            // Allocate for the device_points_within_epsilon:
            errorCode = hipMalloc((int **)&device_points_within_epsilon,sizeof(int));
            if (errorCode != hipSuccess)
            {
                cout<<"Error: Memory allocation on GPU for total points count varaible failed!!!"<<endl;
            }


            // Allocate memory for the pointData array:
            errorCode = hipMalloc((pointData**)&device_data,sizeof(pointData)*N);
            if (errorCode != hipSuccess)
            {
                cout<<"Error: Memory allocation on GPU for PointData array failed!!!"<<endl;
            }


            // Allocate memory for epsilon value on the GPU:
            errorCode = hipMalloc((double**)&device_epsilon,sizeof(double));
            if (errorCode != hipSuccess)
            {
                cout<<"Error: Memory allocation on GPU for device_epsilon failed!!!"<<endl;
            }

            // Memcpy starting timestamp:
            double memcpy_tstart = omp_get_wtime();

            // Copy the info to the device (GPU):            
            errorCode=hipMemcpy( device_points_within_epsilon, host_points_within_epsilon, sizeof(int), hipMemcpyHostToDevice);
            if(errorCode != hipSuccess) 
            {
                cout << "\nError: device_points_within_epsilon memcpy error with code " << errorCode << endl; 
            }

            // Copy device_data
            errorCode=hipMemcpy( device_data, data, sizeof(pointData)*N, hipMemcpyHostToDevice);
            if(errorCode != hipSuccess) 
            {
                cout << "\nError: device_points_within_epsilon memcpy error with code " << errorCode << endl; 
            }

            // Copy epsilon data:
            errorCode=hipMemcpy( device_epsilon, &epsilon, sizeof(double), hipMemcpyHostToDevice);
            if(errorCode != hipSuccess) 
            {
                cout << "\nError: device_epsilon memcpy error with code " << errorCode << endl; 
            }


            // Memcpy ending timestamp:
            double memcpy_tend = omp_get_wtime();



            // Initializing kernel properties:
            const unsigned int totalBlocks = ceil((N*1.0)/(1024.0));
            printf("\nTotal blocks in the kernel: %d",totalBlocks);


            // Getting kernel launch start time:
            double kernel_starttime = omp_get_wtime();

            // Launching the kernel:
            //vectorAdd<<<totalBlocks,1024>>>(dev_A, dev_B, dev_C);
            pointEpsilonBruteForce<<<totalBlocks,1024>>>(device_data,device_points_within_epsilon,device_epsilon);

            errorCode = hipGetLastError();

            if(errorCode != hipSuccess)
            {
		        cout<<"Error after kernel launch "<<errorCode<<endl;
	        }

             // Getting kernel launch start time:
            double kernel_endtime = omp_get_wtime();

            // Copy data from device to host 
            errorCode = hipMemcpy(host_points_within_epsilon, device_points_within_epsilon, sizeof(int), hipMemcpyDeviceToHost);

            double memcpy2_tend = omp_get_wtime();

            if(errorCode != hipSuccess) 
            {
                cout << "\nError: getting result form GPU error with code " << errorCode << endl; 
            }

            // Synchronizing the devices:
            hipDeviceSynchronize();
        
	
    // Overall ending timestamp
	double tend=omp_get_wtime();
	
    printf("\nTotal number of data-points generated: %d",N);
    printf("\nNumber of points within epsilon: %d",*host_points_within_epsilon);
	printf("\nTotal time (s): %f",tend-tstart);
    printf("\nData Transfer (Host-to-Device) Time: %f",memcpy_tend - memcpy_tstart);
    printf("\nData Transfer (Device-to-Host) Time: %f",memcpy2_tend - kernel_endtime);
    printf("\nKernel execution time: %f",kernel_endtime - kernel_starttime);    


	free(data);
	printf("\n");
	return 0;
}


// ************************************************ GPU Kernel Function Defintion ******************************************************************


__global__ void pointEpsilonBruteForce(pointData* all_data_points,int *device_points_within_epsilon,double *device_epsilon)
{

    // Obtaining the threads ID:
    unsigned int tid=threadIdx.x+ (blockIdx.x*blockDim.x);

    // Because, number of threads created
    // might be greater than N:
    if (tid >= N)
    {
	    return;
    }


    // Initializing self pointData object for
    // this thread:
    pointData my_point;
    
    my_point.x = all_data_points[tid].x;
    my_point.y = all_data_points[tid].y;


    // Varaible for keeping
    // points within this thread's
    // pointdata's epsilon range:
    int total_points_within_epsilon = 0;


    // Temporary pointData object:
    pointData temp;

    double distance;

    for(unsigned long int index = 0; index < N; index++)
    {
        // Obtaining the pointData object
        // at index's info:
        temp.x = all_data_points[index].x;
        temp.y = all_data_points[index].y;

        // Calculating the distance:
        distance = sqrt(pow(temp.x - my_point.x,2)+pow(temp.y - my_point.y,2));

        if(distance <= *device_epsilon)
        {
            total_points_within_epsilon++;
        }

    }

    /* Atomically adding the counts to the
    // global variable:
    for(unsigned long int i = 0; i < total_points_within_epsilon; i++)
    {
        atomicAdd(device_points_within_epsilon,1);    
    } */
    atomicAdd(device_points_within_epsilon,total_points_within_epsilon);


    return;

}


// ************************************************************* GPU Warmpup Routine **************************************************


__global__ void warmup(unsigned int * tmp) 
{
    if (threadIdx.x==0)
    {
        *tmp=555;
    }    

    return;
}



void warmUpGPU()
{

    printf("\nWarming up GPU for time trialing...\n");	

    unsigned int * dev_tmp;
    unsigned int * tmp;

    tmp=(unsigned int*)malloc(sizeof(unsigned int));

    *tmp=0;

    hipError_t errorCode = hipSuccess;

    errorCode = hipMalloc((unsigned int**)&dev_tmp, sizeof(unsigned int));

    if(errorCode != hipSuccess) 
    {
        cout << "\nError: dev_tmp error with code " << errorCode << endl; 
    }

    warmup<<<1,256>>>(dev_tmp);

    //copy data from device to host 
    errorCode=hipMemcpy( tmp, dev_tmp, sizeof(unsigned int), hipMemcpyDeviceToHost);
    if(errorCode != hipSuccess) 
    {
        cout << "\nError: getting tmp result form GPU error with code " << errorCode << endl; 
    }

    printf("\ntmp (changed to 555 on GPU): %d",*tmp);

    hipFree(dev_tmp);

    return;
}


//Do not modify the dataset generator or you will get the wrong answer
void generateDataset(struct pointData * data)
{

	//seed RNG
	srand(SEED);


	for (unsigned int i=0; i<N; i++){
		data[i].x=1000.0*((double)(rand()) / RAND_MAX);	
		data[i].y=1000.0*((double)(rand()) / RAND_MAX);	
	}
	

}